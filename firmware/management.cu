#include "management.cuh"


int h_image_width_ = 0;
int h_image_height_ = 0;


#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

dim3 threadsPerBlock(8, 8);
dim3 blocksPerGrid((d_image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
(d_image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);



// int d_image_width_ = 1920;
// int d_image_height_ = 1200;
// bool load_calib_data_flag_ = false;


SystemConfigDataStruct cuda_system_config_settings_machine_;
void cuda_set_param_system_config(SystemConfigDataStruct param)
{
	cuda_system_config_settings_machine_ = param;
}

bool cuda_set_projector_version(int version)
{
    switch (version)
    {
    case DF_PROJECTOR_3010:
    {
		int dlp_width = 1280;
		int dlp_height = 720;
		cuda_set_param_dlp_resolution(dlp_width,dlp_height);
 

        return true;
    }
    break;

    case DF_PROJECTOR_4710:
    {
		int dlp_width = 1920;
		int dlp_height = 1080;
 
		cuda_set_param_dlp_resolution(dlp_width,dlp_height);

 
        return true;
    }
    break;

    default:
        break;
    }

	return false;
}

bool cuda_set_camera_resolution(int width,int height)
{
	h_image_width_ = width;
	h_image_height_ = height;
 
	d_image_width_ = width;
	d_image_height_ = height;

	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &width, sizeof(int)); 
	if(error_code!= hipSuccess)
	{
		return false;
	}
		
	error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &height, sizeof(int));

	if(error_code!= hipSuccess)
	{
		return false;
	}
 

	blocksPerGrid.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x;
	blocksPerGrid.y = (height + threadsPerBlock.y - 1) / threadsPerBlock.y;

	return true;
}


//分配basic内存
bool cuda_malloc_basic_memory()
{
    for (int i = 0; i < MAX_PATTERNS_NUMBER; i++)
    {
        hipMalloc((void **)&d_patterns_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }

    // hipBindTexture(0,texture_patterns_0,d_patterns_list_[0]);
	// hipBindTexture(0,texture_patterns_1,d_patterns_list_[1]);
	// hipBindTexture(0,texture_patterns_2,d_patterns_list_[2]);
	// hipBindTexture(0,texture_patterns_3,d_patterns_list_[3]);
	// hipBindTexture(0,texture_patterns_4,d_patterns_list_[4]);
	// hipBindTexture(0,texture_patterns_5,d_patterns_list_[5]);
	// hipBindTexture(0,texture_patterns_6,d_patterns_list_[6]);
	// hipBindTexture(0,texture_patterns_7,d_patterns_list_[7]);
	// hipBindTexture(0,texture_patterns_8,d_patterns_list_[8]);
	// hipBindTexture(0,texture_patterns_9,d_patterns_list_[9]);
	// hipBindTexture(0,texture_patterns_10,d_patterns_list_[10]);
	// hipBindTexture(0,texture_patterns_11,d_patterns_list_[11]);
	// hipBindTexture(0,texture_patterns_12,d_patterns_list_[12]);
	// hipBindTexture(0,texture_patterns_13,d_patterns_list_[13]);
	// hipBindTexture(0,texture_patterns_14,d_patterns_list_[14]);
	// hipBindTexture(0,texture_patterns_15,d_patterns_list_[15]);
	// hipBindTexture(0,texture_patterns_16,d_patterns_list_[16]);
	// hipBindTexture(0,texture_patterns_17,d_patterns_list_[17]);
	// hipBindTexture(0,texture_patterns_18,d_patterns_list_[18]);

 

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_wrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_unwrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	} 

	hipMalloc((void**)&d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	hipMalloc((void**)&d_mask_map_, d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	hipMalloc((void**)&d_fisher_mask_, d_image_height_ * d_image_width_ * sizeof(unsigned char));


	hipMalloc((void**)&d_camera_intrinsic_, 3*3 * sizeof(float));
	hipMalloc((void**)&d_project_intrinsic_, 3 * 3 * sizeof(float));

	hipMalloc((void**)&d_camera_distortion_, 1* 5 * sizeof(float));
	hipMalloc((void**)&d_projector_distortion_, 1 * 5 * sizeof(float));

	hipMalloc((void**)&d_rotation_matrix_, 3 * 3 * sizeof(float));
	hipMalloc((void**)&d_translation_matrix_, 1 * 3 * sizeof(float));


	hipMalloc((void**)&d_fisher_confidence_map, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_point_cloud_map_, 3*d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_triangulation_error_map_, d_image_height_*d_image_width_ * sizeof(float));
 
 	hipMalloc((void**)&d_single_pattern_mapping_, 4000*2000 * sizeof(float)); 
	hipMalloc((void**)&d_single_pattern_minimapping_, 128*128 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

    LOG(INFO)<<"d_image_width_: "<<d_image_width_;
    LOG(INFO)<<"d_image_height_: "<<d_image_height_;
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_basic_memory()
{

	for (int i = 0; i< MAX_PATTERNS_NUMBER; i++)
	{  
		hipFree(d_patterns_list_[i]); 
	}

	// hipUnbindTexture(texture_patterns_0);
	// hipUnbindTexture(texture_patterns_1);
	// hipUnbindTexture(texture_patterns_2);
	// hipUnbindTexture(texture_patterns_3);
	// hipUnbindTexture(texture_patterns_4);
	// hipUnbindTexture(texture_patterns_5);
	// hipUnbindTexture(texture_patterns_6);
	// hipUnbindTexture(texture_patterns_7);
	// hipUnbindTexture(texture_patterns_8);
	// hipUnbindTexture(texture_patterns_9);
	// hipUnbindTexture(texture_patterns_10);
	// hipUnbindTexture(texture_patterns_11);
	// hipUnbindTexture(texture_patterns_12);
	// hipUnbindTexture(texture_patterns_13);
	// hipUnbindTexture(texture_patterns_14);
	// hipUnbindTexture(texture_patterns_15);
	// hipUnbindTexture(texture_patterns_16);
	// hipUnbindTexture(texture_patterns_17);
	// hipUnbindTexture(texture_patterns_18);

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{  
		hipFree(d_wrap_map_list_[i]);
		hipFree(d_confidence_map_list_[i]); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{ 
		hipFree(d_unwrap_map_list_[i]); 
	}

	hipFree(d_fisher_confidence_map);
	hipFree(d_fisher_mask_);
    hipFree(d_mask_map_);
    hipFree(d_brightness_map_);
    hipFree(d_point_cloud_map_);
    hipFree(d_depth_map_);
    hipFree(d_triangulation_error_map_);

    hipFree(d_camera_intrinsic_);
	hipFree(d_project_intrinsic_); 
	hipFree(d_camera_distortion_);
	hipFree(d_projector_distortion_); 
	hipFree(d_rotation_matrix_);
	hipFree(d_translation_matrix_);
 
    hipFree(d_single_pattern_mapping_);
    hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
 

	return true;
}

 //分配hdr内存
bool cuda_malloc_hdr_memory()
{
	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{
		hipMalloc((void**)&d_hdr_depth_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_hdr_brightness_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
		hipMalloc((void**)&d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_hdr_memory()
{
    for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{ 
		hipFree(d_hdr_depth_map_list_[i]);
		hipFree(d_hdr_brightness_list_[i]);
		hipFree(d_hdr_bright_pixel_sum_list_[i]);
	}
	
	return true;
}

//分配repetition内存
bool cuda_malloc_repetition_memory()
{
	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipMalloc((void**)&d_repetition_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipMalloc((void**)&d_repetition_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
 
 	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipMalloc((void**)&d_repetition_02_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_repetition_memory()
{

	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipFree(d_repetition_patterns_list_[i]); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipFree(d_repetition_merge_patterns_list_[i]);  
	}

	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipFree(d_repetition_02_merge_patterns_list_[i]);  
	}
	
	return true;
}


/********************************************************************************************/
//copy 
void cuda_copy_calib_data(float* camera_intrinsic, float* project_intrinsic, float* camera_distortion,
	float* projector_distortion, float* rotation_matrix, float* translation_matrix)
{
  
	CHECK(hipMemcpy(d_camera_intrinsic_, camera_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_project_intrinsic_, project_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_camera_distortion_, camera_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_projector_distortion_, projector_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_rotation_matrix_, rotation_matrix, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_translation_matrix_, translation_matrix, 1* 3 * sizeof(float), hipMemcpyHostToDevice));

	load_calib_data_flag_ = 1;

 
}

void cuda_copy_talbe_to_memory(float* mapping,float* mini_mapping,float* rotate_x,float* rotate_y,float* r_1,float base_line)
{
   
	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3*3 * sizeof(float), hipMemcpyHostToDevice)); 
	CHECK(hipMemcpyAsync(d_single_pattern_minimapping_, mini_mapping, 128 * 128 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_single_pattern_mapping_, mapping, 4000*2000 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	
    d_baseline_ = base_line;  
 

	LOG(INFO)<<"d_baseline_: "<<d_baseline_;
	hipDeviceSynchronize();
}


bool cuda_copy_pattern_to_memory(unsigned char* pattern_ptr,int serial_flag)
{
	if(serial_flag> MAX_PATTERNS_NUMBER)
	{
		return false;
	}

	CHECK(hipMemcpyAsync(d_patterns_list_[serial_flag], pattern_ptr, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

void cuda_copy_pointcloud_from_memory(float* pointcloud)
{ 
	CHECK(hipMemcpy(pointcloud, d_point_cloud_map_, 3 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
}

void cuda_copy_depth_from_memory(float* depth)
{
	CHECK(hipMemcpy(depth, d_depth_map_, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
} 

void cuda_copy_brightness_from_memory(unsigned char* brightness)
{
	CHECK(hipMemcpy(brightness, d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost)); 
}

void cuda_copy_brightness_to_memory(unsigned char* brightness)
{ 
	CHECK(hipMemcpyAsync(d_brightness_map_, brightness, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

/********************************************************************************************/


bool cuda_compute_phase_shift(int serial_flag)
{
	 
	switch(serial_flag)
	{
		case 0:
		{ 
        	LOG(INFO)<<"kernel_four_step_phase_shift:"<<d_image_width_;
			int i= 0;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);

				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 1:
		{

			int i= 4;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
			
		}
		break;
		case 2:
		{ 
			int i= 8;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 3:
		{ 
			int i= 12; 
			kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3],d_patterns_list_[i + 4],d_patterns_list_[i + 5] ,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
 
            
				// cuda_six_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> > (d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				// hipDeviceSynchronize();

				// cv::Mat phase(1200, 1920, CV_32F, cv::Scalar(0));
				// CHECK(hipMemcpy(phase.data, d_wrap_map_list_[serial_flag], 1 * image_height_ * image_width_ * sizeof(float), hipMemcpyDeviceToHost));
				// cv::imwrite("phase1.tiff",phase);
		}
		break;
		case 4:
		{
			int i= 18;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 5:
		{
			int i= 22;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 6:
		{
			int i= 26;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
  
		default :
			break;
	}

	
	
	return true;
}


bool cuda_normalize_phase(int serial_flag)
{
    switch(serial_flag)
	{ 
        case 0:
		{   
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], (float)128.0, d_unwrap_map_list_[0]);  
		}
		break; 
		case 1:
		{   
  
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)18., d_unwrap_map_list_[1]); 
		}
		break;

		case 2:
		{ 
			kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)72., d_unwrap_map_list_[1]); 
		}
		break;

		default :
			break;
	}


	return true;
}

bool cuda_unwrap_phase_shift(int serial_flag)
{

	switch(serial_flag)
	{ 
		case 1:
		{  
            kernel_unwrap_variable_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[0], d_wrap_map_list_[1], 8.0, CV_PI, d_unwrap_map_list_[0]);
  
		}
		break;

		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[2], 4.0,CV_PI, d_unwrap_map_list_[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[3], 4.0,1.5, d_unwrap_map_list_[0]); 
 
		}
		break;
		case 4:
		{
 
		}
		break;
		case 5:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[4], d_wrap_map_list_[5], 8.0,CV_PI, d_unwrap_map_list_[1]);
		}
		break;
		case 6:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[6], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
			LOG(INFO)<<"unwrap 6:  ";

		}
		break;
		case 7:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[7], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
		 	LOG(INFO)<<"unwrap 7:  ";

		}
		break;
 

		default :
			break;
	}


	return true;
}

bool cuda_unwrap_phase_shift_base_fisher_confidence(int serial_flag)
{

	switch(serial_flag)
	{ 
		case 1:
		{  
            kernel_unwrap_variable_phase_base_confidence<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[0], d_wrap_map_list_[1], 8.0, CV_PI, FISHER_RATE_1, d_fisher_confidence_map, d_unwrap_map_list_[0]);
  
		}
		break;

		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase_base_confidence << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[2], 4.0,CV_PI, FISHER_RATE_2, d_fisher_confidence_map, d_unwrap_map_list_[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase_base_confidence << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[3], 4.0,1.5, FISHER_RATE_3, d_fisher_confidence_map, d_unwrap_map_list_[0]); 
 
		}
		break;
		default :
			break;
	}


	return true;
}

/********************************************************************************************************************************************/

bool cuda_generate_pointcloud_base_minitable()
{
		if(1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_reflect_filter)
	{ 
		LOG(INFO)<<"filter_reflect_noise start:"; 
		cuda_filter_reflect_noise(d_unwrap_map_list_[0]); 

		hipDeviceSynchronize();
		LOG(INFO)<<"filter_reflect_noise end";
	}

	kernel_reconstruct_pointcloud_base_minitable<< <blocksPerGrid, threadsPerBlock>> > (d_image_width_,d_image_height_,d_xL_rotate_x_,d_xL_rotate_y_,d_single_pattern_minimapping_,d_R_1_,d_baseline_,
	d_confidence_map_list_[3],d_unwrap_map_list_[0],d_point_cloud_map_,d_depth_map_);

 
}


bool cuda_generate_pointcloud_base_table()
{
	// cv::Mat phase(2048,2448,CV_32FC1,cv::Scalar(0));
	// CHECK(hipMemcpy(phase.data, d_unwrap_map_list_[0], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// cv::imwrite("phase.tiff", phase);
	
	// if(1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_reflect_filter)
	// { 
	// 	LOG(INFO)<<"filter_reflect_noise start:"; 
	// 	cuda_filter_reflect_noise(d_unwrap_map_list_[0]); 
	//
	// 	hipDeviceSynchronize();
	// 	LOG(INFO)<<"filter_reflect_noise end";
	// }

	kernel_reconstruct_pointcloud_base_table << <blocksPerGrid, threadsPerBlock>> > (d_image_width_,d_image_height_,d_xL_rotate_x_,d_xL_rotate_y_,d_single_pattern_mapping_,d_R_1_,d_baseline_,
	d_confidence_map_list_[3],d_unwrap_map_list_[0],d_point_cloud_map_,d_depth_map_);

	// cv::Mat depth(2048,2448,CV_32FC1,cv::Scalar(0));
	// CHECK(hipMemcpy(depth.data, d_depth_map_, 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// cv::imwrite("depth.tiff", depth);
}

/********************************************************************************************************************************************/

bool cuda_copy_result_to_hdr(int serial_flag,int brigntness_serial)
{
	if(!load_calib_data_flag_)
	{
		return false;
	}
 

	CHECK(hipMemcpyAsync(d_hdr_depth_map_list_[serial_flag], d_depth_map_, 1 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToDevice)); 
	CHECK(hipMemcpyAsync(d_hdr_brightness_list_[serial_flag], d_patterns_list_[brigntness_serial], 1 * d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToDevice));

	float val  = 0;
	CHECK(hipMemcpyAsync(d_hdr_bright_pixel_sum_list_[serial_flag], &val, sizeof(float), hipMemcpyHostToDevice)); 
 	cuda_count_sum_pixel << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_list_[serial_flag],d_image_height_,d_image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);
 
	LOG(INFO)<<"cuda_copy_result_to_hdr: "<<serial_flag;
	return true;
}


bool cuda_merge_hdr_data(int hdr_num,float* depth_map, unsigned char* brightness)
{
	
	LOG(INFO)<<"sum pixels ";
	float sum_pixels_list[6];  

    for(int i= 0;i<hdr_num;i++)
    { 
		CHECK(hipMemcpy(&sum_pixels_list[i], d_hdr_bright_pixel_sum_list_[i], 1* sizeof(float), hipMemcpyDeviceToHost));
    }
 
 
	std::vector<float> param_list;
	std::vector<int> id; 
	std::vector<bool> flag_list;

	for (int i = 0; i < hdr_num; i++)
	{ 
        param_list.push_back(sum_pixels_list[i]);
		id.push_back(0);
		flag_list.push_back(true);
    } 
   	std::sort(param_list.begin(),param_list.end(),std::greater<float>());
 
 
	for (int i = 0; i < hdr_num; i++)
	{ 
		
		for(int j= 0;j< hdr_num;j++)
		{
			if(param_list[i] == sum_pixels_list[j])
			{
				if(flag_list[j])
				{ 
					id[i] = j;
					flag_list[j] = false; 
					break;
				}
			}
		}
		 
    } 

 
	for (int i = 0; i < hdr_num; i++)
	{ 
        LOG(INFO)<<"sum pixels "<<i<<": "<<sum_pixels_list[i]<<" _ "<<id[i];
    }
 

	switch(hdr_num)
	{
		case 1:
		{

			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_list_[0], 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[0], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		} 
		break;
		case 2:
		{
			cuda_merge_hdr_2 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);

				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 3:
		{
			cuda_merge_hdr_3 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 4:
		{
			cuda_merge_hdr_4 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],d_hdr_depth_map_list_[id[3]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 5:
		{
			cuda_merge_hdr_5 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 6:
		{
			cuda_merge_hdr_6 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],d_hdr_depth_map_list_[id[5]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				 d_hdr_brightness_list_[id[5]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;

		default:
		 		return false;

	}

 	// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[id[0]], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
 	CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[hdr_num-1], 1*h_image_height_*h_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
	LOG(INFO)<<"DHR Finished!";

	return true;
}
/********************************************************************************************************************************************/

bool cuda_copy_repetition_pattern_to_memory(unsigned char* patterns_ptr,int serial_flag)
{
	CHECK(hipMemcpyAsync(d_repetition_patterns_list_[serial_flag], patterns_ptr, h_image_height_*h_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice));
}

bool cuda_merge_repetition_patterns(int repetition_serial)
{

	int merge_serial = repetition_serial%6; 
	kernel_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_patterns_list_[repetition_serial],h_image_height_, h_image_width_,d_repetition_merge_patterns_list_[merge_serial]);

	return true;
}


bool cuda_compute_merge_phase(int repetition_count)
{

	kernel_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_merge_patterns_list_[0], d_repetition_merge_patterns_list_[1],
		d_repetition_merge_patterns_list_[2],d_repetition_merge_patterns_list_[3],d_repetition_merge_patterns_list_[4],d_repetition_merge_patterns_list_[5] ,
		repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	return true;
}


bool cuda_clear_repetition_02_patterns()
{
	for(int i = 0;i< D_REPETITION_02_MAX_NUM;i++)
	{ 
		 hipMemset(d_repetition_02_merge_patterns_list_[i], 0,h_image_height_*h_image_width_*sizeof(ushort));
		// CHECK(hipMemcpyAsync(d_repetition_02_merge_patterns_list_[i], &val,image_width_* image_height_*sizeof(ushort), hipMemcpyHostToDevice));
	}
	// hipDeviceSynchronize();
  
  return true;
}

bool cuda_merge_repetition_02_patterns(int repetition_serial)
{
	// int merge_serial = repetition_serial%19; 
	kernel_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_patterns_list_[repetition_serial],h_image_height_, h_image_width_,d_repetition_02_merge_patterns_list_[repetition_serial]);

	return true;
}


bool cuda_compute_merge_repetition_02_phase(int repetition_count,int phase_num)
{
	
	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],
		d_repetition_02_merge_patterns_list_[2],d_repetition_02_merge_patterns_list_[3],repetition_count, h_image_height_, h_image_width_,d_wrap_map_list_[0], d_confidence_map_list_[0]);
			
	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5],
		d_repetition_02_merge_patterns_list_[6],d_repetition_02_merge_patterns_list_[7],repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[1], d_confidence_map_list_[1]);

	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[8], d_repetition_02_merge_patterns_list_[9],
		d_repetition_02_merge_patterns_list_[10],d_repetition_02_merge_patterns_list_[11],repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[2], d_confidence_map_list_[2]);
	
	kernel_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[12], d_repetition_02_merge_patterns_list_[13],
		d_repetition_02_merge_patterns_list_[14],d_repetition_02_merge_patterns_list_[15],d_repetition_02_merge_patterns_list_[16],d_repetition_02_merge_patterns_list_[17] ,
		repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	if(1 == phase_num)
	{
		kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_02_merge_patterns_list_[18],repetition_count,h_image_height_, h_image_width_,d_brightness_map_);
	}
	else if (2 == phase_num)
	{

		int i = 18;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count, h_image_height_, h_image_width_,d_wrap_map_list_[4], d_confidence_map_list_[4]);

		i = 22;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[5], d_confidence_map_list_[5]);

		i = 26;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[6], d_confidence_map_list_[6]);

		i = 30;
		kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			  d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], d_repetition_02_merge_patterns_list_[i + 4], d_repetition_02_merge_patterns_list_[i + 5],
																			  repetition_count, h_image_height_, h_image_width_, d_wrap_map_list_[7], d_confidence_map_list_[7]);

		kernel_merge_brigntness_map<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[36], repetition_count, h_image_height_, h_image_width_,d_brightness_map_);

		 
	}

	return true;
}
/********************************************************************************************************************************************/
//filter
void cuda_remove_points_base_radius_filter(float dot_spacing,float radius,int threshold_num)
{
	LOG(INFO)<<"remove_base_radius_filter start:"; 
 
	// //相机像素为5.4um、焦距12mm。dot_spacing = 5.4*distance/12000 mm，典型值0.54mm（1200） 

	float d2 = dot_spacing*dot_spacing;
	float r2 = radius*radius;
	
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_filter_radius_outlier_removal), hipFuncCachePreferL1);
	kernel_filter_radius_outlier_removal << <blocksPerGrid, threadsPerBlock >> > (h_image_height_,h_image_width_,d_point_cloud_map_,d_mask_map_,d2,r2,threshold_num); 
	
	hipDeviceSynchronize();

	// cv::Mat mask(1200, 1920, CV_8U, cv::Scalar(0));
	// CHECK(hipMemcpy(mask.data, d_mask_map_, 1 * h_image_height_ * h_image_width_ * sizeof(uchar), hipMemcpyDeviceToHost));
	// cv::imwrite("mask.bmp", mask);
	LOG(INFO)<<"remove start:";
	kernel_removal_points_base_mask << <blocksPerGrid, threadsPerBlock >> > (h_image_height_,h_image_width_,d_point_cloud_map_,d_depth_map_,d_mask_map_); 

    hipDeviceSynchronize();
 

	LOG(INFO)<<"remove_base_radius_filter finished!";
}


void cuda_filter_reflect_noise(float * const unwrap_map)
{
    // dim3 threadsPerBlock_p(img_width);
    // dim3 blocksPerGrid_p(img_height);

	//按行来组织线程
    dim3 threadsPerBlock_p(4, 4);
    // dim3 blocksPerGrid_p(15,2);
    dim3 blocksPerGrid_p;
	if(1200 == h_image_height_)
	{
		blocksPerGrid_p.x = (40 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (30 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	else if(2048 == h_image_height_)
	{
		blocksPerGrid_p.x = (64 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (32 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}

 
 	kernel_filter_reflect_noise << <blocksPerGrid_p, threadsPerBlock_p >> > ( h_image_height_,h_image_width_, unwrap_map);
}


void fisher_filter(float fisher_confidence_val)
{
	//按行来组织线程
    dim3 threadsPerBlock_p(4, 4);
    dim3 blocksPerGrid_p;
	if(1200 == h_image_height_)
	{
		blocksPerGrid_p.x = (40 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (30 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	else if(2048 == h_image_height_)
	{
		blocksPerGrid_p.x = (64 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (32 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	LOG(INFO)<<"fisher start"; 
	kernel_fisher_filter <<< blocksPerGrid_p, threadsPerBlock_p >>> (h_image_height_, h_image_width_, (FISHER_CENTER_LOW + (fisher_confidence_val * FISHER_CENTER_RATE)), d_fisher_confidence_map, d_fisher_mask_, d_unwrap_map_list_[0]);//
	hipDeviceSynchronize();
	LOG(INFO)<<"fisher end"; 
}



/*****************************************************************************************************************************************************/
//repetition

void cuda_copy_phase_from_cuda_memory(float* phase_x,float* phase_y)
{
	CHECK(hipMemcpy(phase_x, d_unwrap_map_list_[0], d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
	CHECK(hipMemcpy(phase_y, d_unwrap_map_list_[1], d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
}




/*****************************************************************************************************************************************************/









