#include "memory_management.cuh"
#include "encode.cuh"
#include "reconstruct.cuh"

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

dim3 threadsPerBlock(8, 8);
dim3 blocksPerGrid((d_image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
(d_image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);



// int d_image_width_ = 1920;
// int d_image_height_ = 1200;
// bool load_calib_data_flag_ = false;

bool cuda_set_camera_version(int version)
{
    switch (version)
    {
    case DFX_800:
    {
		int dlp_width = 1280;
		int dlp_height = 720;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));
  
		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));

        return true;
    }
    break;

    case DFX_1800:
    {
		int dlp_width = 1920;
		int dlp_height = 1080;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));

		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));
        return true;
    }
    break;

    default:
        break;
    }

	return false;
}

bool cuda_set_camera_resolution(int width,int height)
{
	d_image_width_ = width;
	d_image_height_ = height;
 

	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &width, sizeof(int)); 
	if(error_code!= hipSuccess)
	{
		return false;
	}
		
	error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &height, sizeof(int));

	if(error_code!= hipSuccess)
	{
		return false;
	}
 

	blocksPerGrid.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x;
	blocksPerGrid.y = (height + threadsPerBlock.y - 1) / threadsPerBlock.y;

	return true;
}


//分配basic内存
bool cuda_malloc_basic_memory()
{
    for (int i = 0; i < MAX_PATTERNS_NUMBER; i++)
    {
        hipMalloc((void **)&d_patterns_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }

    // hipBindTexture(0,texture_patterns_0,d_patterns_list_[0]);
	// hipBindTexture(0,texture_patterns_1,d_patterns_list_[1]);
	// hipBindTexture(0,texture_patterns_2,d_patterns_list_[2]);
	// hipBindTexture(0,texture_patterns_3,d_patterns_list_[3]);
	// hipBindTexture(0,texture_patterns_4,d_patterns_list_[4]);
	// hipBindTexture(0,texture_patterns_5,d_patterns_list_[5]);
	// hipBindTexture(0,texture_patterns_6,d_patterns_list_[6]);
	// hipBindTexture(0,texture_patterns_7,d_patterns_list_[7]);
	// hipBindTexture(0,texture_patterns_8,d_patterns_list_[8]);
	// hipBindTexture(0,texture_patterns_9,d_patterns_list_[9]);
	// hipBindTexture(0,texture_patterns_10,d_patterns_list_[10]);
	// hipBindTexture(0,texture_patterns_11,d_patterns_list_[11]);
	// hipBindTexture(0,texture_patterns_12,d_patterns_list_[12]);
	// hipBindTexture(0,texture_patterns_13,d_patterns_list_[13]);
	// hipBindTexture(0,texture_patterns_14,d_patterns_list_[14]);
	// hipBindTexture(0,texture_patterns_15,d_patterns_list_[15]);
	// hipBindTexture(0,texture_patterns_16,d_patterns_list_[16]);
	// hipBindTexture(0,texture_patterns_17,d_patterns_list_[17]);
	// hipBindTexture(0,texture_patterns_18,d_patterns_list_[18]);

 

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_wrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_unwrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}
  
	hipMalloc((void**)&d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char)); 


	hipMalloc((void**)&d_camera_intrinsic_, 3*3 * sizeof(float));
	hipMalloc((void**)&d_project_intrinsic_, 3 * 3 * sizeof(float));

	hipMalloc((void**)&d_camera_distortion_, 1* 5 * sizeof(float));
	hipMalloc((void**)&d_projector_distortion_, 1 * 5 * sizeof(float));

	hipMalloc((void**)&d_rotation_matrix_, 3 * 3 * sizeof(float));
	hipMalloc((void**)&d_translation_matrix_, 1 * 3 * sizeof(float));


	hipMalloc((void**)&d_point_cloud_map_, 3*d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_triangulation_error_map_, d_image_height_*d_image_width_ * sizeof(float));
 
 	hipMalloc((void**)&d_single_pattern_mapping_, 4000*2000 * sizeof(float)); 
	hipMalloc((void**)&d_single_pattern_minimapping_, 128*128 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

    LOG(INFO)<<"d_image_width_: "<<d_image_width_;
    LOG(INFO)<<"d_image_height_: "<<d_image_height_;
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_basic_memory()
{

	for (int i = 0; i< MAX_PATTERNS_NUMBER; i++)
	{  
		hipFree(d_patterns_list_[i]); 
	}

	// hipUnbindTexture(texture_patterns_0);
	// hipUnbindTexture(texture_patterns_1);
	// hipUnbindTexture(texture_patterns_2);
	// hipUnbindTexture(texture_patterns_3);
	// hipUnbindTexture(texture_patterns_4);
	// hipUnbindTexture(texture_patterns_5);
	// hipUnbindTexture(texture_patterns_6);
	// hipUnbindTexture(texture_patterns_7);
	// hipUnbindTexture(texture_patterns_8);
	// hipUnbindTexture(texture_patterns_9);
	// hipUnbindTexture(texture_patterns_10);
	// hipUnbindTexture(texture_patterns_11);
	// hipUnbindTexture(texture_patterns_12);
	// hipUnbindTexture(texture_patterns_13);
	// hipUnbindTexture(texture_patterns_14);
	// hipUnbindTexture(texture_patterns_15);
	// hipUnbindTexture(texture_patterns_16);
	// hipUnbindTexture(texture_patterns_17);
	// hipUnbindTexture(texture_patterns_18);

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{  
		hipFree(d_wrap_map_list_[i]);
		hipFree(d_confidence_map_list_[i]); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{ 
		hipFree(d_unwrap_map_list_[i]); 
	}

    hipFree(d_brightness_map_);
    hipFree(d_point_cloud_map_);
    hipFree(d_depth_map_);
    hipFree(d_triangulation_error_map_);

    hipFree(d_camera_intrinsic_);
	hipFree(d_project_intrinsic_); 
	hipFree(d_camera_distortion_);
	hipFree(d_projector_distortion_); 
	hipFree(d_rotation_matrix_);
	hipFree(d_translation_matrix_);
 
    hipFree(d_single_pattern_mapping_);
    hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
 

	return true;
}

 //分配hdr内存
bool cuda_malloc_hdr_memory()
{
	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{
		hipMalloc((void**)&d_hdr_depth_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_hdr_brightness_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
		hipMalloc((void**)&d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_hdr_memory()
{
    for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{ 
		hipFree(d_hdr_depth_map_list_[i]);
		hipFree(d_hdr_brightness_list_[i]);
		hipFree(d_hdr_bright_pixel_sum_list_[i]);
	}
	
	return true;
}

//分配repetition内存
bool cuda_malloc_repetition_memory()
{
	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipMalloc((void**)&d_repetition_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipMalloc((void**)&d_repetition_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
 
 	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipMalloc((void**)&d_repetition_02_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_repetition_memory()
{

	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipFree(d_repetition_patterns_list_[i]); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipFree(d_repetition_merge_patterns_list_[i]);  
	}

	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipFree(d_repetition_02_merge_patterns_list_[i]);  
	}
	
	return true;
}


/********************************************************************************************/
//copy 
void cuda_copy_calib_data(float* camera_intrinsic, float* project_intrinsic, float* camera_distortion,
	float* projector_distortion, float* rotation_matrix, float* translation_matrix)
{
  
	CHECK(hipMemcpy(d_camera_intrinsic_, camera_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_project_intrinsic_, project_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_camera_distortion_, camera_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_projector_distortion_, projector_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_rotation_matrix_, rotation_matrix, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_translation_matrix_, translation_matrix, 1* 3 * sizeof(float), hipMemcpyHostToDevice));

	load_calib_data_flag_ = 1;

 
}

void cuda_copy_talbe_to_memory(float* mapping,float* mini_mapping,float* rotate_x,float* rotate_y,float* r_1,float base_line)
{
   
	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3*3 * sizeof(float), hipMemcpyHostToDevice)); 
	CHECK(hipMemcpyAsync(d_single_pattern_minimapping_, mini_mapping, 128 * 128 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_single_pattern_mapping_, mapping, 4000*2000 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	
    d_baseline_ = base_line;  
 

	LOG(INFO)<<"d_baseline_: "<<d_baseline_;
	hipDeviceSynchronize();
}


bool cuda_copy_pattern_to_memory(unsigned char* pattern_ptr,int serial_flag)
{
	if(serial_flag>= MAX_PATTERNS_NUMBER)
	{
		return false;
	}

	CHECK(hipMemcpyAsync(d_patterns_list_[serial_flag], pattern_ptr, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

void cuda_copy_pointcloud_from_memory(float* pointcloud)
{ 
	CHECK(hipMemcpy(pointcloud, d_point_cloud_map_, 3 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
}

void cuda_copy_depth_from_memory(float* depth)
{
	CHECK(hipMemcpy(depth, d_depth_map_, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
} 

void cuda_copy_brightness_from_memory(unsigned char* brightness)
{
	CHECK(hipMemcpy(brightness, d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost)); 
}

void cuda_copy_brightness_to_memory(unsigned char* brightness)
{ 
	CHECK(hipMemcpyAsync(d_brightness_map_, brightness, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

/********************************************************************************************/


bool cuda_compute_phase_shift(int serial_flag)
{
	 
	switch(serial_flag)
	{
		case 0:
		{ 
        	LOG(INFO)<<"kernel_four_step_phase_shift:"<<d_image_width_;
			int i= 0;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);

				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 1:
		{

			int i= 4;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
			
		}
		break;
		case 2:
		{ 
			int i= 8;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 3:
		{ 
			int i= 12; 
			kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3],d_patterns_list_[i + 4],d_patterns_list_[i + 5] ,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
 
            
				// cuda_six_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> > (d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				// hipDeviceSynchronize();

				// cv::Mat phase(1200, 1920, CV_32F, cv::Scalar(0));
				// CHECK(hipMemcpy(phase.data, d_wrap_map_list_[serial_flag], 1 * image_height_ * image_width_ * sizeof(float), hipMemcpyDeviceToHost));
				// cv::imwrite("phase1.tiff",phase);
		}
		break;
		case 4:
		{
			int i= 18;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 5:
		{
			int i= 22;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 6:
		{
			int i= 26;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
  
		default :
			break;
	}

	
	
	return true;
}


bool cuda_normalize_phase(int serial_flag)
{
    switch(serial_flag)
	{ 
        case 0:
		{   
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], (float)128.0, d_unwrap_map_list_[0]);  
		}
		break; 
		case 1:
		{   
  
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)18., d_unwrap_map_list_[1]); 
		}
		break;

		case 2:
		{ 
			kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)72., d_unwrap_map_list_[1]); 
		}
		break;

		default :
			break;
	}


	return true;
}

bool cuda_unwrap_phase_shift(int serial_flag)
{

	switch(serial_flag)
	{ 
		case 1:
		{  
            kernel_unwrap_variable_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[0], d_wrap_map_list_[1], 8.0, CV_PI, d_unwrap_map_list_[0]);
  
		}
		break;

		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[2], 4.0,CV_PI, d_unwrap_map_list_[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[3], 4.0,1.5, d_unwrap_map_list_[0]); 
 
		}
		break;
		case 4:
		{
 
		}
		break;
		case 5:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[4], d_wrap_map_list_[5], 8.0,CV_PI, d_unwrap_map_list_[1]);
		}
		break;
		case 6:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[6], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
			LOG(INFO)<<"unwrap 6:  ";

		}
		break;
		case 7:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[7], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
		 	LOG(INFO)<<"unwrap 7:  ";

		}
		break;
 

		default :
			break;
	}


	return true;
}

/********************************************************************************************************************************************/

bool cuda_generate_pointcloud_base_table()
{
	cv::Mat phase(2048,2448,CV_32FC1,cv::Scalar(0));
	CHECK(hipMemcpy(phase.data, d_unwrap_map_list_[0], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	cv::imwrite("phase.tiff", phase);

	kernel_reconstruct_pointcloud_base_table << <blocksPerGrid, threadsPerBlock>> > (d_image_width_,d_image_height_,d_xL_rotate_x_,d_xL_rotate_y_,d_single_pattern_mapping_,d_R_1_,d_baseline_,
	d_confidence_map_list_[3],d_unwrap_map_list_[0],d_point_cloud_map_,d_depth_map_);

	cv::Mat depth(2048,2448,CV_32FC1,cv::Scalar(0));
	CHECK(hipMemcpy(depth.data, d_depth_map_, 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	cv::imwrite("depth.tiff", depth);
}

/********************************************************************************************************************************************/