#include "memory_management.cuh"
#include "easylogging++.h"

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


// int d_image_width_ = 1920;
// int d_image_height_ = 1200;
// bool load_calib_data_flag_ = false;

bool cuda_set_camera_version(int version)
{
    switch (version)
    {
    case DFX_800:
    {
		int dlp_width = 1280;
		int dlp_height = 720;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));
  
		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));

        return true;
    }
    break;

    case DFX_1800:
    {
		int dlp_width = 1920;
		int dlp_height = 1080;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));

		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));
        return true;
    }
    break;

    default:
        break;
    }

	return false;
}

bool cuda_set_camera_resolution(int width,int height)
{
	d_image_width_ = width;
	d_image_height_ = height;
 

	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &width, sizeof(int)); 
	if(error_code!= hipSuccess)
	{
		return false;
	}
		
	error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &height, sizeof(int));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	return true;
}


//分配basic内存
bool cuda_malloc_basic_memory()
{
    for (int i = 0; i < MAX_PATTERNS_NUMBER; i++)
    {
        hipMalloc((void **)&d_patterns_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }

    // hipBindTexture(0,texture_patterns_0,d_patterns_list_[0]);
	// hipBindTexture(0,texture_patterns_1,d_patterns_list_[1]);
	// hipBindTexture(0,texture_patterns_2,d_patterns_list_[2]);
	// hipBindTexture(0,texture_patterns_3,d_patterns_list_[3]);
	// hipBindTexture(0,texture_patterns_4,d_patterns_list_[4]);
	// hipBindTexture(0,texture_patterns_5,d_patterns_list_[5]);
	// hipBindTexture(0,texture_patterns_6,d_patterns_list_[6]);
	// hipBindTexture(0,texture_patterns_7,d_patterns_list_[7]);
	// hipBindTexture(0,texture_patterns_8,d_patterns_list_[8]);
	// hipBindTexture(0,texture_patterns_9,d_patterns_list_[9]);
	// hipBindTexture(0,texture_patterns_10,d_patterns_list_[10]);
	// hipBindTexture(0,texture_patterns_11,d_patterns_list_[11]);
	// hipBindTexture(0,texture_patterns_12,d_patterns_list_[12]);
	// hipBindTexture(0,texture_patterns_13,d_patterns_list_[13]);
	// hipBindTexture(0,texture_patterns_14,d_patterns_list_[14]);
	// hipBindTexture(0,texture_patterns_15,d_patterns_list_[15]);
	// hipBindTexture(0,texture_patterns_16,d_patterns_list_[16]);
	// hipBindTexture(0,texture_patterns_17,d_patterns_list_[17]);
	// hipBindTexture(0,texture_patterns_18,d_patterns_list_[18]);

 

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_wrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_unwrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}
  
	hipMalloc((void**)&d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char)); 


	hipMalloc((void**)&d_camera_intrinsic_, 3*3 * sizeof(float));
	hipMalloc((void**)&d_project_intrinsic_, 3 * 3 * sizeof(float));

	hipMalloc((void**)&d_camera_distortion_, 1* 5 * sizeof(float));
	hipMalloc((void**)&d_projector_distortion_, 1 * 5 * sizeof(float));

	hipMalloc((void**)&d_rotation_matrix_, 3 * 3 * sizeof(float));
	hipMalloc((void**)&d_translation_matrix_, 1 * 3 * sizeof(float));


	hipMalloc((void**)&d_point_cloud_map_, 3*d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_triangulation_error_map_, d_image_height_*d_image_width_ * sizeof(float));
 
 	hipMalloc((void**)&d_single_pattern_mapping_, 4000*2000 * sizeof(float)); 
	hipMalloc((void**)&d_single_pattern_minimapping_, 128*128 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

    LOG(INFO)<<"d_image_width_: "<<d_image_width_;
    LOG(INFO)<<"d_image_height_: "<<d_image_height_;
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_basic_memory()
{

	for (int i = 0; i< MAX_PATTERNS_NUMBER; i++)
	{  
		hipFree(d_patterns_list_[i]); 
	}

	// hipUnbindTexture(texture_patterns_0);
	// hipUnbindTexture(texture_patterns_1);
	// hipUnbindTexture(texture_patterns_2);
	// hipUnbindTexture(texture_patterns_3);
	// hipUnbindTexture(texture_patterns_4);
	// hipUnbindTexture(texture_patterns_5);
	// hipUnbindTexture(texture_patterns_6);
	// hipUnbindTexture(texture_patterns_7);
	// hipUnbindTexture(texture_patterns_8);
	// hipUnbindTexture(texture_patterns_9);
	// hipUnbindTexture(texture_patterns_10);
	// hipUnbindTexture(texture_patterns_11);
	// hipUnbindTexture(texture_patterns_12);
	// hipUnbindTexture(texture_patterns_13);
	// hipUnbindTexture(texture_patterns_14);
	// hipUnbindTexture(texture_patterns_15);
	// hipUnbindTexture(texture_patterns_16);
	// hipUnbindTexture(texture_patterns_17);
	// hipUnbindTexture(texture_patterns_18);

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{  
		hipFree(d_wrap_map_list_[i]);
		hipFree(d_confidence_map_list_[i]); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{ 
		hipFree(d_unwrap_map_list_[i]); 
	}

    hipFree(d_brightness_map_);
    hipFree(d_point_cloud_map_);
    hipFree(d_depth_map_);
    hipFree(d_triangulation_error_map_);

    hipFree(d_camera_intrinsic_);
	hipFree(d_project_intrinsic_); 
	hipFree(d_camera_distortion_);
	hipFree(d_projector_distortion_); 
	hipFree(d_rotation_matrix_);
	hipFree(d_translation_matrix_);
 
    hipFree(d_single_pattern_mapping_);
    hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
 

	return true;
}

 //分配hdr内存
bool cuda_malloc_hdr_memory()
{
	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{
		hipMalloc((void**)&d_hdr_depth_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_hdr_brightness_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
		hipMalloc((void**)&d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_hdr_memory()
{
    for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{ 
		hipFree(d_hdr_depth_map_list_[i]);
		hipFree(d_hdr_brightness_list_[i]);
		hipFree(d_hdr_bright_pixel_sum_list_[i]);
	}
	
	return true;
}

//分配repetition内存
bool cuda_malloc_repetition_memory()
{
	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipMalloc((void**)&d_repetition_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipMalloc((void**)&d_repetition_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
 
 	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipMalloc((void**)&d_repetition_02_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_repetition_memory()
{

	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipFree(d_repetition_patterns_list_[i]); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipFree(d_repetition_merge_patterns_list_[i]);  
	}

	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipFree(d_repetition_02_merge_patterns_list_[i]);  
	}
	
	return true;
}


/********************************************************************************************/
//copy 
void cuda_copy_calib_data(float* camera_intrinsic, float* project_intrinsic, float* camera_distortion,
	float* projector_distortion, float* rotation_matrix, float* translation_matrix)
{
  
	CHECK(hipMemcpy(d_camera_intrinsic_, camera_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_project_intrinsic_, project_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_camera_distortion_, camera_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_projector_distortion_, projector_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_rotation_matrix_, rotation_matrix, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_translation_matrix_, translation_matrix, 1* 3 * sizeof(float), hipMemcpyHostToDevice));

	load_calib_data_flag_ = 1;

 
}

void cuda_copy_talbe_to_memory(float* mapping,float* mini_mapping,float* rotate_x,float* rotate_y,float* r_1,float base_line)
{
   
	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3*3 * sizeof(float), hipMemcpyHostToDevice)); 
	CHECK(hipMemcpyAsync(d_single_pattern_minimapping_, mini_mapping, 128 * 128 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_single_pattern_mapping_, mapping, 4000*2000 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	
    d_baseline_ = base_line;  

	LOG(INFO)<<"d_baseline_: "<<d_baseline_;
	hipDeviceSynchronize();
}






/********************************************************************************************/